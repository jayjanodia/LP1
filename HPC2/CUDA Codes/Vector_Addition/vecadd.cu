
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <chrono>

using namespace std;
using namespace std::chrono;

__global__ void addVectors(int *a, int *b, int *c, int n){
	int start = blockIdx.x * 256;

	for(int i = start; i < min(start+256, n); i++){
		c[i] = a[i] + b[i];
	}	

}
int main(){
	int n;
	cout << "Enter number of elements: " ;
	cin >> n;

	int *arr1 = new int[n];
	int *arr2 = new int[n];
	int *res = new int[n];

	int *serialAdd = new int[n];
	srand(time(0));

	for(int i = 0; i < n; i++){
		arr1[i] = rand() % 100;
		arr2[i] = rand() % 100;
	}

	int *devA, *devB, *devRes;

	hipMalloc(&devA, n * sizeof(int));
	hipMalloc(&devB, n * sizeof(int));
	hipMalloc(&devRes, n * sizeof(int));

	hipMemcpy(devA, arr1, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devB, arr2, n * sizeof(int), hipMemcpyHostToDevice);

	auto start = high_resolution_clock::now();

	int blocks = ceil(n * 1.0f/256.0f);
	
	addVectors<<<blocks,1>>>(devA, devB, devRes,n);
	auto stop = high_resolution_clock::now();

	cout << "Time for parallel execution: " << duration_cast<microseconds>(stop-start).count() << endl;
	
	hipMemcpy(res, devRes, n * sizeof(int), hipMemcpyDeviceToHost);
	
	start = high_resolution_clock::now();
	for(int i = 0; i < n; i++){
		serialAdd[i] = arr1[i] + arr2[i];
	}
	stop = high_resolution_clock::now();
	cout << "Time for serial execution: " << duration_cast<microseconds>(stop-start).count() << endl;

	for(int i = 0; i < n; i++){
		if(res[i] != serialAdd[i]){
			cout << "Wrong answer at: " << i << endl;
      return 1;
		}
	}

  cout << "AC" << endl;
	return 0;
}
