
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<chrono>
using namespace std;
using namespace std::chrono;

/*__global__ void addition(int *a, int *b, int *c, int n) {
    int large_id = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = large_id; i < min(large_id + 256, n); i++) {
        c[i] = a[i] + b[i];
        printf("Test 1 2 3");
    }
}*/

__global__ void addition(int *a, int *b, int *c, int n) {
	int large_id = blockIdx.x * blockDim.x + threadIdx.x;
	while (large_id < n) {
	//if(large_id < n) {
		c[large_id] = a[large_id] + b[large_id];
		large_id += blockDim.x*gridDim.x;
	}
}

void addition_cpu(int *a, int *b, int *c, int n) {
    for(int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

int main(void) {
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;
    int n;
    cin>>n;
    cout<<n;
    a = (int *) malloc(n * sizeof(int));
    b = (int *) malloc(n * sizeof(int));
    c = (int *) malloc(n * sizeof(int));

    for(int i = 0; i < n; i++) {
        a[i] = i + 1;
        b[i] = i + 1;
        c[i] = 0;
    }

    int size = n * sizeof(int);
    hipMalloc(&dev_a, size);
    hipMalloc(&dev_b, size);
    hipMalloc(&dev_c, size);

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    int blocks, threads;
    blocks = threads = ceil(n * 1.0f/256.0f);
    auto start = high_resolution_clock::now();
    addition <<<blocks, threads>>> (dev_a, dev_b, dev_c, n);
    auto stop = high_resolution_clock::now();
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
    cout<<"\nFor GPU:"<<endl;
    /*for(int i = 0; i < n; i++) {
		cout<<a[i]<<"+"<<b[i]<<"="<<c[i]<<endl;
    }*/
    //verify that gpu did work
	int count = 0;
	bool success = true;
	for(int i = 0; i < n; i++) {
		if((a[i] + b[i]) != c[i]) {
			cout<<"Error in "<<a[i]<<"+"<<b[i]<<"="<<c[i]<<endl;
			success = false;
			count++;
		}
	}
	if (success) cout<<"We did it"<<endl;
    cout<<"Number of errors: "<<count<<endl;
    cout<<"\nTime taken for parallel processing: "<<duration_cast <microseconds> (stop - start).count()<<endl;
    for(int i =0; i < n; i++) {
        c[i] = 0;
    }
    start = high_resolution_clock::now();
    addition_cpu(a, b, c, n);
    stop = high_resolution_clock::now();
    cout<<"For CPU: "<<endl;
    cout<<"\nTime taken for serial processing"<<duration_cast <microseconds> (stop - start).count() <<endl;
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}