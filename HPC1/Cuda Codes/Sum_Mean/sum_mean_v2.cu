
#include <hip/hip_runtime.h>
#include<iostream>
#include<chrono>
using namespace std;
using namespace std::chrono;

__global__ void sum_mean(int *a, int *b, int n) {
	int large_id = blockIdx.x * blockDim.x + threadIdx.x;
	int sum = 0;
	for(int i = large_id; i < min(large_id+ 256, n); i++) {
		sum += a[i];
	}
	b[large_id] = sum;
}

void sum_mean_cpu(int *a, int *b, int n) {
	int sum = 0;
	for(int i = 0; i < n; i++) {
		sum += a[i];
	}
	b[0] = sum;
}

int main(void) {
	int *a, *b, n;
	int *dev_a, *dev_b;
	cout<<"Enter number of elements in array: "<<endl;
	cin>>n;

	//a = new int[n];
	//b = new int[1];
	a = (int *)malloc(n * sizeof(int));
	b = (int *)malloc(sizeof(int));

	for(int i = 0; i < n; i++) {
		a[i] = i + 1;
	}
	/*cout<<"The numbers stored in the array are: "<<endl;
	for(int i = 0; i < n; i++) {
		cout<<a[i] << " ";
	}
	cout<<endl;*/

	hipMalloc(&dev_a, n * sizeof(int));
	hipMalloc(&dev_b, sizeof(int));

	hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(int), hipMemcpyHostToDevice);

	int blocks, threads;
	blocks = threads = ceil(n * 1.0f/256.0f);
	
	auto start = high_resolution_clock::now();
	sum_mean <<<blocks, threads>>> (dev_a, dev_b, n);
	auto stop = high_resolution_clock::now();
	cout<<"For GPU: "<<endl;
	hipMemcpy(b, dev_b, sizeof(int), hipMemcpyDeviceToHost);
	cout<<"The sum is: " << b[0] << "\nThe mean is: " << b[0] / n << "\nThe time taken for parallel execution is: " << duration_cast<microseconds>(stop-start).count() << endl;

	b[0] = 0;
	start = high_resolution_clock::now();
	sum_mean_cpu (a, b, n);
	stop = high_resolution_clock::now();
	cout<<"For CPU: "<<endl;
	cout<<"The sum is: " << b[0] << "\nThe mean is: " << b[0] / n << "\nThe time taken for serial execution is: " << duration_cast<microseconds>(stop-start).count() << endl;
}