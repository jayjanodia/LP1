
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<chrono>

using namespace std;
using namespace std::chrono;

//standard deviation formula: (sqrt(summation(x - mean)^2)/n)

__global__ void standard_deviation(int *a, float *b, float mean, int n) {
    int large_id = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0;
    for(int i = large_id; i < min(large_id + 256, n); i++) {
        sum += (a[i] - mean) * (a[i] - mean);
        //printf("Sum: %f\t a[%d]: %d\n", sum, i, a[i]);
    }
    b[large_id] = sum/n;
}

void standard_deviation_cpu(int *a, float *b, float mean, int n) {
    for(int i = 0; i < n; i++) {
        b[0] += (a[i] - mean) * (a[i] - mean);
    }
    b[0] /=n;
}

int main(void) {
    int *a, *dev_a, n;
    float *b, *dev_b, mean;
    cout<<"Enter number of elements in array: "<<endl;
    cin>>n;
    a = (int *)malloc(n * sizeof(int)); //a = new int[n];
    b = (float *) malloc (sizeof(int)); // b = new float[n];
    //cout<<"The input numbers are: "<<endl;
    for(int i = 0; i < n; i++) {
        a[i] = i + 1;
        //cout<<a[i]<<"\t";
    }
    cout<<endl;

    mean = (n + 1)/2;
    cout<<"Mean: "<<mean<<endl;

    hipMalloc(&dev_a, n * sizeof(int));
    hipMalloc(&dev_b, sizeof(float));
    
    int blocks, threads;
    blocks = threads = ceil(n * 1.0f/256.0f);
    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    auto start = high_resolution_clock::now();
    standard_deviation <<<blocks, threads>>> (dev_a, dev_b, mean, n);
    auto stop = high_resolution_clock::now();

    cout<<"For GPU: "<<endl;
    hipMemcpy(b, dev_b, sizeof(float), hipMemcpyDeviceToHost);
    cout<<"Standard deviation is: "<< sqrt(b[0]) <<"\nTime taken for parallel execution is: "<<duration_cast <microseconds> (stop - start).count() <<endl;

    b[0] = 0.0;
    cout<<"For CPU:" <<endl;
    start = high_resolution_clock::now();
    standard_deviation_cpu(a, b, mean, n);
    stop = high_resolution_clock::now();
    cout<<"Standard deviation is  "<<sqrt(b[0]) <<"\nTime taken for serial execution is: "<< duration_cast <microseconds> (stop - start).count()<<endl;
}