
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void minimum(int *a, int *b, int n) {
    int tid = threadIdx.x;
    int min_limit = 99999;

    for(int i = tid; i < min(tid + 256, n); i++) {
    //for(int i = 0; i < n; i++) will work just fine
        if (min_limit > a[i]) {
            min_limit = a[i];
        }
        //printf("Min limit in %d = %d\n", i,min_limit); //debugging purpose
    }
    b[tid] = min_limit;
}

int main() {
    cout << "Enter the size of the array" << endl;
    int n;
    cin >> n;
    //int a[n]; //does not work in some cuda versions
    int *a = (int *)malloc(n * sizeof(int));
    for(int i = 0; i < n; i++) {
        //a[i] = i;
        a[i] = rand();
    }
    //checking the values of a[i] to see what has been given in the input
    cout<<"The input values given are:"<<endl;
    for(int i = 0; i < n; i++) {
        cout<<a[i]<<"\t";
    }
    cout<<endl;
    int *dev_a, *dev_b;
    int size = n * sizeof(int);

    hipMalloc(&dev_a, size);
    hipMalloc(&dev_b, sizeof(int));

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    minimum<<<1, n>>>(dev_a, dev_b, n);

    int *ans = (int *)malloc(sizeof(int));
    hipMemcpy(ans, dev_b, sizeof(int), hipMemcpyDeviceToHost);
    cout<<"The minimum element is: "<<ans[0]<<endl;
}