
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<chrono>
using namespace std;
using namespace std::chrono;

__global__ void minimum(int *a, int *b, int n) {
    int min_limit = 999999;
    int large_id = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i = large_id; i < min(large_id + 256, n); i++) {
        if (min_limit > a[i]) {
            min_limit = a[i];
        }
        //printf("Min limit for %d is %d\n", i, min_limit);
    }
    b[0] = min_limit;
}
    
void minimum_cpu(int *a, int *b, int n) {
    int min_limit = 9999999;
    for(int i = 0; i < n; i++) {
        if(min_limit > a[i]) {
            min_limit = a[i];
        }
    }
    b[0] = min_limit;
}

int main(void) {
    int *a, *b, n;
    int *dev_a, *dev_b;

    cout<<"Enter the number of elements in the array:"<<endl;
    cin>>n;

    a = (int *) malloc(n * sizeof(int));
    b = (int *) malloc(sizeof(int));

    for(int i = 0; i < n; i++) {
        a[i] = rand();
    }
    for(int i = 0; i < n; i++) {
        cout<< a[i] <<endl;
    }
    hipMalloc(&dev_a, n * sizeof(int));
    hipMalloc(&dev_b, sizeof(int));

    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    
    int blocks, threads;
    blocks = threads = ceil(n * 1.0f/256.0f);

    auto start = high_resolution_clock::now();
    minimum <<<blocks, threads>>> (dev_a, dev_b, n);
    auto stop = high_resolution_clock::now();

    hipMemcpy(b, dev_b, sizeof(int), hipMemcpyDeviceToHost);
    cout<<"For GPU: "<<endl;
    cout<<"Minimum value is: "<<b[0] << "\nTime taken for parallel execution: "<< duration_cast <microseconds> (stop - start).count() <<endl;

    b[0] = 0;
    start = high_resolution_clock::now();
    minimum_cpu(a, b, n);
    stop = high_resolution_clock::now();

    cout<<"For CPU: "<<endl;
    cout<<"Minimum value is: "<<b[0] << "\nTime taken for serial execution: "<< duration_cast <microseconds> (stop - start).count() << endl;
}