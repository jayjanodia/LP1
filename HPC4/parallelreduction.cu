#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;
using namespace std::chrono;
__global__ void sum(int *arr, int *sumArr, int n){
	int start = blockIdx.x * 256;

	int sum = 0;
	for(int i = start; i < min(start+256,n); i++){
		sum += arr[i];
	}

	sumArr[blockIdx.x] = sum;
}

__global__ void pMin(int *arr, int *minArr, int n){
	int start = blockIdx.x * 256;
	int minVal = 9999999;
	for(int i = start; i < min(start+256,n); i++){
		if(arr[i] < minVal){minVal = arr[i];}
	}
	minArr[blockIdx.x] = minVal;
}

//These 2 functions are for standard deviation

__global__ void standardDeviation(float *arr, float* calcArr, float mean, int n){
	int start = blockIdx.x * 256;

	float aggregate = 0;
	for(int i = start; i < min(start + 256, n); i++){
		aggregate = aggregate + ((arr[i] - mean) *(arr[i] - mean));
	}

	calcArr[blockIdx.x] = aggregate;
}

__global__ void addAll(float *arr, float *calcArr, int n){
	int start = blockIdx.x * 256;

	float sum = 0;
	for(int i = start; i < min(start + 256, n); i++){
		sum += arr[i];
	}

	calcArr[blockIdx.x] = sum;
}

__global__ void pMax(float *arr, float *maxArr, int n){
	int start = blockIdx.x * 256;

	int maxm = -9999999;
	for(int i = start; i < min(start+256,n); i++){
		if(arr[i] > maxm){
			maxm = arr[i];
		}
	}
	maxArr[blockIdx.x] = maxm;
}



int main(){

	int n;
	cout << "Enter number of elements: ";
	cin >> n;

	int *hostArr,*devArr,*devSumArr, *devMinArr;

	hostArr = new int[n];
	for(int i = 0; i < n; i++){
		hostArr[i] = i+1;
	}

	hipMalloc(&devArr, n * 4);
	int blocks = ceil(n * 1.0f/256.0f);

	hipMalloc(&devSumArr, blocks * 4);
	
	
	hipMemcpy(devArr, hostArr, n*4, hipMemcpyHostToDevice);
	//Sum
	int curr = n;
  auto start = high_resolution_clock::now();
	while(curr > 1){
		sum<<<blocks, 1>>>(devArr, devSumArr, curr);
		hipMemcpy(devArr,devSumArr,blocks*4,hipMemcpyDeviceToDevice);
		curr = ceil(curr* 1.0f/256.0f);
		blocks = ceil(curr*1.0f/256.0f);
				
	}
  auto stop = high_resolution_clock::now();
  
  cout << "parallel: " << duration_cast<microseconds>(stop - start).count() << endl;
  
  start = high_resolution_clock::now();
  long myVal = 0;
  for(int i = 0; i < n; i++){
    myVal = myVal + hostArr[i];
  }
  stop = high_resolution_clock::now();
  cout << "serial: " << duration_cast<microseconds>(stop - start).count() << endl;
	int sum;
	hipMemcpy(&sum,devArr,4,hipMemcpyDeviceToHost);
	cout << "Sum: " << sum << endl;
	
	//Min value i
	hipMalloc(&devMinArr, blocks * 4);
	//Refill device array with values of host array
	hipMemcpy(devArr,hostArr,n*4,hipMemcpyHostToDevice);
	
	curr = n;
	blocks = ceil(n * 1.0f/256.0f);
	while(curr > 1){
		pMin<<<blocks, 1>>>(devArr, devMinArr, curr);
		hipMemcpy(devArr, devMinArr, blocks*4, hipMemcpyDeviceToDevice);
		curr = blocks;
		blocks = ceil(curr*1.0f/256.0f);
	
	}
	int minVal;
	hipMemcpy(&minVal, devArr, 4, hipMemcpyDeviceToHost);
	cout << "min val: " << minVal << endl;

	float mean = (float)sum/n;

	//Standard deviation
	float *fDevArr, *fStdSum;
	blocks = ceil(n * 1.0f/ 256.0f);

	//Create an aggregate array
	hipMalloc(&fStdSum, sizeof(float) * blocks);
	//Copy mean's value to gpu mean
	
	float *floatHost = new float[n];

	for(int i = 0; i < n; i++){
		floatHost[i] = (float)hostArr[i];
	}
	//Allocate device array space in gpu
	hipMalloc(&fDevArr, sizeof(float) * n);
	//Refill device array with values of host array
	hipMemcpy(fDevArr,floatHost, 4 * n, hipMemcpyHostToDevice);
	
	standardDeviation<<<blocks, 1>>>(fDevArr, fStdSum, mean, n);

  float *myArr = new float[blocks];
  hipMemcpy(myArr, fStdSum, sizeof(float) * blocks, hipMemcpyDeviceToHost);
  
  float total = 0;
  for(int i = 0; i < blocks; i++){
    total += myArr[i];
  }
  
  total /= n;
  total = sqrt(total);
  cout << "validation standard deviation: " << total << endl;
	curr = blocks;
	while(curr > 1){
		hipMemcpy(fDevArr, fStdSum, curr * sizeof(float), hipMemcpyDeviceToDevice);
		blocks = ceil(curr * 1.0f/256.0f);
    cout << "blocks for aggregation: " << blocks << endl;
		addAll<<<blocks, 1>>>(fDevArr,fStdSum, curr);
		curr = blocks;
	}
	
	float stdDeviation;
	hipMemcpy(&stdDeviation, fStdSum, sizeof(float), hipMemcpyDeviceToHost);

	stdDeviation /= n;
	stdDeviation = sqrt(stdDeviation);

	cout << "Standard deviation: " << stdDeviation << endl;

	float *intermediateMax;
	blocks = ceil(n * 1.0f/256.0f);
	hipMalloc(&intermediateMax, blocks * sizeof(float));

	
	hipMemcpy(fDevArr,floatHost, 4 * n, hipMemcpyHostToDevice);
	curr = n;
	while(curr > 1){
		pMax<<<blocks, 1>>>(fDevArr, intermediateMax, curr);
		hipMemcpy(fDevArr, intermediateMax, blocks*sizeof(float), hipMemcpyDeviceToDevice);
		float *tempArr = new float[blocks];
		hipMemcpy(tempArr, intermediateMax, blocks*sizeof(float), hipMemcpyDeviceToHost);
		cout << "Intermediate maximum values: ";
		for(int i = 0; i < blocks; i++){
			cout << tempArr[i] << " ";
		}
		cout << endl;
		
		curr = blocks;
		blocks = ceil(curr * 1.0f/256.0f);
	}

	float maxm = 0;
	hipMemcpy(&maxm, intermediateMax, sizeof(float), hipMemcpyDeviceToHost);

	cout << "Maximum: " << maxm << endl;	
}	
